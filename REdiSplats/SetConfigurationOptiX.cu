#include "Header.cuh"

// *************************************************************************************************

bool SetConfigurationOptiX(SOptiXRenderConfig& config_OptiX) {
	hipMemcpyToSymbol(HIP_SYMBOL(bg_color_R), &config_OptiX.bg_color_R, sizeof(float));
	bg_color_R_host = config_OptiX.bg_color_R;
	hipMemcpyToSymbol(HIP_SYMBOL(bg_color_G), &config_OptiX.bg_color_G, sizeof(float));
	bg_color_G_host = config_OptiX.bg_color_G;
	hipMemcpyToSymbol(HIP_SYMBOL(bg_color_B), &bg_color_B, sizeof(float));
	bg_color_B_host = config_OptiX.bg_color_B;

	number_of_sides_host = config_OptiX.number_of_sides;

	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH0), &config_OptiX.lr_SH0, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH0_exponential_decay_coefficient), &config_OptiX.lr_SH0_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH0_final), &config_OptiX.lr_SH0_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(SH1_activation_iter), &config_OptiX.SH1_activation_iter, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH1), &config_OptiX.lr_SH1, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH1_exponential_decay_coefficient), &config_OptiX.lr_SH1_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH1_final), &config_OptiX.lr_SH1_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(SH2_activation_iter), &config_OptiX.SH2_activation_iter, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH2), &config_OptiX.lr_SH2, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH2_exponential_decay_coefficient), &config_OptiX.lr_SH2_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH2_final), &config_OptiX.lr_SH2_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(SH3_activation_iter), &config_OptiX.SH3_activation_iter, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH3), &config_OptiX.lr_SH3, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH3_exponential_decay_coefficient), &config_OptiX.lr_SH3_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH3_final), &config_OptiX.lr_SH3_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(SH4_activation_iter), &config_OptiX.SH4_activation_iter, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH4), &config_OptiX.lr_SH4, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH4_exponential_decay_coefficient), &config_OptiX.lr_SH4_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_SH4_final), &config_OptiX.lr_SH4_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(lr_alpha), &config_OptiX.lr_alpha, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_alpha_exponential_decay_coefficient), &config_OptiX.lr_alpha_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_alpha_final), &config_OptiX.lr_alpha_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(lr_m), &config_OptiX.lr_m, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_m_exponential_decay_coefficient), &config_OptiX.lr_m_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_m_final), &config_OptiX.lr_m_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(lr_s), &config_OptiX.lr_s, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_s_exponential_decay_coefficient), &config_OptiX.lr_s_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_s_final), &config_OptiX.lr_s_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(lr_q), &config_OptiX.lr_q, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_q_exponential_decay_coefficient), &config_OptiX.lr_q_exponential_decay_coefficient, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lr_q_final), &config_OptiX.lr_q_final, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(densification_frequency), &config_OptiX.densification_frequency, sizeof(int));
	densification_frequency_host = config_OptiX.densification_frequency;

	hipMemcpyToSymbol(HIP_SYMBOL(densification_start_epoch), &config_OptiX.densification_start_epoch, sizeof(int));
	densification_start_epoch_host = config_OptiX.densification_start_epoch;

	hipMemcpyToSymbol(HIP_SYMBOL(densification_end_epoch), &config_OptiX.densification_end_epoch, sizeof(int));
	densification_end_epoch_host = config_OptiX.densification_end_epoch;

	hipMemcpyToSymbol(HIP_SYMBOL(alpha_threshold_for_Gauss_removal), &config_OptiX.alpha_threshold_for_Gauss_removal, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(min_s_coefficients_clipping_threshold), &config_OptiX.min_s_coefficients_clipping_threshold, sizeof(float));
	min_s_coefficients_clipping_threshold_host = config_OptiX.min_s_coefficients_clipping_threshold;

	hipMemcpyToSymbol(HIP_SYMBOL(max_s_coefficients_clipping_threshold), &config_OptiX.max_s_coefficients_clipping_threshold, sizeof(float));
	max_s_coefficients_clipping_threshold_host = config_OptiX.max_s_coefficients_clipping_threshold;

	hipMemcpyToSymbol(HIP_SYMBOL(min_s_norm_threshold_for_Gauss_removal), &config_OptiX.min_s_norm_threshold_for_Gauss_removal, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(max_s_norm_threshold_for_Gauss_removal), &config_OptiX.max_s_norm_threshold_for_Gauss_removal, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(mu_grad_norm_threshold_for_densification), &config_OptiX.mu_grad_norm_threshold_for_densification, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(s_norm_threshold_for_split_strategy), &config_OptiX.s_norm_threshold_for_split_strategy, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(split_ratio), &config_OptiX.split_ratio, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(lambda), &config_OptiX.lambda, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(ray_termination_T_threshold), &config_OptiX.ray_termination_T_threshold, sizeof(float));
	ray_termination_T_threshold_host = config_OptiX.ray_termination_T_threshold;

	last_significant_Gauss_alpha_gradient_precision_host = config_OptiX.last_significant_Gauss_alpha_gradient_precision;

	hipMemcpyToSymbol(HIP_SYMBOL(chi_square_squared_radius), &config_OptiX.chi_square_squared_radius, sizeof(float));
	chi_square_squared_radius_host = config_OptiX.chi_square_squared_radius;

	hipMemcpyToSymbol(HIP_SYMBOL(max_Gaussians_per_ray), &config_OptiX.max_Gaussians_per_ray, sizeof(int));
	max_Gaussians_per_ray_host = config_OptiX.max_Gaussians_per_ray;

	hipMemcpyToSymbol(HIP_SYMBOL(max_Gaussians_per_model), &config_OptiX.max_Gaussians_per_model, sizeof(int));
	max_Gaussians_per_model_host = config_OptiX.max_Gaussians_per_model;

	tmp_arrays_growth_factor_host = config_OptiX.tmp_arrays_growth_factor;

	return true;
}