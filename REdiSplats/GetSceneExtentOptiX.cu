#include "Header.cuh"

// *************************************************************************************************

bool GetSceneExtentOptiX(float &scene_extent_host) {
	hipError_t error_CUDA;

	error_CUDA = hipMemcpyFromSymbol(&scene_extent_host, HIP_SYMBOL(scene_extent), sizeof(float));
	if (error_CUDA != hipSuccess) goto Error;

	return true;
Error:
	return false;
}